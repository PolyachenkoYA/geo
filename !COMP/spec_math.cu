#include "hip/hip_runtime.h"
#include "spec_math.cuh"

int rightV(double3* v, double3 n, double3 v0)
{
	int i, res = -1;
	double cs, max_cs = -2, kv = 1/length(v0);

	// reflected ray have the least angle with v0 |=> the beggest cos(v_new,v0)
	for(i = 0; i < 4; ++i){
		if(dot(v0, n)*dot(v[i], n) < 0){ // exclude other side of surface
			cs = dot(v[i], v0)/length(v[i])*kv;
			if(cs > max_cs){ // determine reflected ray
				res = i;
				max_cs = cs;
			}
		}
	}

	return res;
}

double3 newV(double3 n, double3 v_old, double3 vx, double3 vy)
{
	double3 v_news[4];
	// we don't know the direction of n and nr, so we have to check all 4 possible variants
	// to choose the one that really is the physicaly reflected ray
	v_news[0] = vx + vy;
	v_news[1] = vx - vy;
	v_news[2] = -v_news[1];
	v_news[3] = -v_news[0];

	int i = rightV(v_news, n, v_old);
	return i == -1 ? make_double3(0,0,0) : v_news[i];
}
