#include "hip/hip_runtime.h"
#include "general_math.cuh"

bool isNan(double3 v)
{
	return std::isnan(v.x) || std::isnan(v.y) || std::isnan(v.z);
}

char bool2int(bool b){ return b ? 1 : 0; }

int3 get_Nslc(double3 X, double3 dX){ return make_int3(int(X.x / dX.x), int(X.y / dX.y), int(X.z / dX.z)); }
int ind3D_to_ind(int3 I, int3 Sz)
{
	return I.x + Sz.x * (I.y + I.z * Sz.y);
}

bool is0(double3 v){ return (v.x == 0) && (v.y == 0) && (v.z == 0); }

double myRnd(void){ return rand()/double(RAND_MAX); }
double myRnd(double a, double b){
	if(a>b) swap(a,b);
	return myRnd()*(b-a)+a;
}
double3 vecByAngles(double phi, double tht )
{
	double ct = cos(tht);
	return make_double3(ct * cos(phi), ct * sin(phi), sin(tht));
}
double3 rndVec(double V)
{
	return vecByAngles(myRnd(-pi, pi), myRnd(-pi_d2, pi_d2))*V;
}
// this gauss is checked - it's really gauss
double gaussFnc(double x, double sgm, double x0)
{
	double b = (x-x0)/sgm;
	return exp(-b*b/2) / (sqrt(2*M_PI)*sgm);
}
double gaussRand(double sgm, double x0, double rng)
{
	rng *= sgm;
	double x;
	double y0 = 1.0/(sqrt(2*M_PI)*sgm); //y0 = gaussFnc(x0, sgm, x0); // max value
	double xl = x0-rng, xr = x0+rng;

	do{
		x = myRnd(xl, xr);
	}while(myRnd(0, y0) > gaussFnc(x, sgm, x0));

	return x;
}

bool almostEq(double a, double b, double _eps)
{
	return (b == 0 ? std::abs(a) : std::abs(a/b-1)) < _eps;
}
double epsDlt(double a, double b, double _eps)
{
	return b == 0 ? (a == 0 ? 0 : 1/_eps) : (std::abs(a) > std::abs(b) ? (a/b-1) : (b/a-1));
}
double float_part(double x)
{
    return x - int(x);
}

double cos_sin(double x)
{
    return sqrtf(1-x*x);
}

double sqr(double x)
{
	return x*x;
}

int sgn(double x, double _eps)
{
    if(x >= _eps){
    	return 1;
    } else if(x <= -_eps){
        return -1;
    } else {
        return 0;
    }
}
