#include "hip/hip_runtime.h"
#include "format.cuh"

void time_progress(time_t real_start_t, time_t curr_t, double done_part, string proc_name)
{
	time_t real_t = curr_t - real_start_t;
	double left_t = real_t * (1/done_part - 1);
	int b_i = int(left_t);
	time_t b_t = curr_t + b_i;

	cout << proc_name << " " << 100*done_part << " %          \n"
		 << "time used " << real_t/3600 << ":" << (real_t%3600)/60 << ":" << real_t%60 << "          \n"
		 << "time left " << b_i/3600 << ":" << (b_i%3600)/60 << ":" << b_i%60 << "          \n"
		 << "last save: " << string(ctime(&curr_t))
		 << "finish   : " << string(ctime(&b_t))
		 << "\r"      // goto begin & erase
		 << "\033[A"  // up & erase
		 << "\033[A"  // up & erase
		 << "\033[A"  // up & erase
		 << "\033[A"  // up & erase
		 << "\033[A"; // up & erase
	/*
		 * At first sight it seems nothing will be printed because I print and erase all right away.
		 * But I suppose that in some case \smth affect output-buffer but doesn't cause any screen.reprint()
		 * so everything (empty strings from previous cout) is printed in the beginning of next time print when actual chars are printed
		 * So in fact user sees everything erased just before new information to be printed
		 */
}

template<typename T>
//int stp(T str)
void stp(T str)
{
	cerr << str << endl;
	cin.get();
	//int r;
	//cin >> r;
	//return r;
}

template<typename T>
string vectorToStr(vector<T> v, string sp)
{
        string s="";
        for(int i = 0; i < v.size()-1; ++i) s+= (toString(v[i])+sp);
        return s+toString(v[v.size()-1]);
}
template<typename T>
void printVector(ostream &output, vector<T> v, string sp1, string sp2, string sp3)
{
	int sz = v.size();
    output << sp1;
    for(int i = 0; i < sz-1; ++i) output << v[i] << sp2;
    output << v[sz-1] << sp3;
}
template<typename T>
T sumVector(vector<T> v)
{
        T s=0;
        for(int i = 0; i < v.size(); ++i) s+=v[i];
        return s;
}
template <typename T>
string toString(T val)
{
    std::ostringstream oss;
    oss << val;
    return oss.str();
}
template<typename T>
T fromString(const string& s)
{
  std::istringstream iss(s);
  T res;
  iss >> res;
  return res;
}

string toLower(string s)
{
	char d = 'a'-'A';
	for(int i = 0; i<s.size(); ++i) if((s[i]>='A') && (s[i]<='Z')) s[i]+=d;
	return s;
}
string toUpper(string s)
{
	char d = 'A'-'a';
	for(int i = 0; i<s.size(); ++i) if((s[i]>='a') && (s[i]<='z')) s[i]+=d;
	return s;
}
void printX3(ostream &output, double3 r, string sp1, string sp2, string sp3){ output << toStr(r, sp1, sp2, sp3); }
void printX3(ostream &output, int3 r, string sp1, string sp2, string sp3){ output << toStr(r, sp1, sp2, sp3); }

int CHECK(int n)
{
    if(n){
    	ofstream Fout(global_logFname.c_str(), ios::app);
    	//if(!Fout){ return CANT_OPEN_FILE_FOR_WRITING; }
    	if(!Fout){ return n; }

    	if(n == SAY_IT){
    		Fout << error_handl_string;
    	} else {
    		string _s = "error #" + toString(n) + "                                                 \n" +
    				    "message:                                                \n" +
    				    error_handl_string + "\n";
    		Fout << "\n" << _s << "\n";
    		cout << _s;
    	}
    	Fout.close();
    }

    return n;
}

template <typename T>
void SAY_LOG(T s)
{
	error_handl_string = toString(s);
	CHECK(SAY_IT);
}

string toStr(double3 d, string sp1, string sp2, string sp3){ return sp1 + toString(d.x) + sp2 + toString(d.y) + sp2 + toString(d.z) + sp3; }
string toStr(int3 d, string sp1, string sp2, string sp3){ return sp1 + toString(d.x) + sp2 + toString(d.y) + sp2 + toString(d.z) + sp3; }

vector<double> d3ToV(double3 v){
	double vp[3] = {v.x, v.y, v.z};
	vector<double> vv;
	vv.assign(vp, vp+3);
	return vv;
}
