#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : comp_geo.cu
 Author      : PolyachYA
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include "../../geo.cuh"

int main(int argc, char **argv)
{

	if(argc >= 2){
		// --------------------------------------------
		// ------------ handle input ------------------
		// --------------------------------------------
		time_t now = time(0);
		string sessionID = string(ctime(&now));
		Params prm;
		prm.model_name = string(argv[1]);
		prm.prm_filename = prm.model_name + ".prm";
		prm.tet_filename = prm.model_name + ".tet";
		prm.material_filename = prm.model_name + ".mat";
		global_logFname = prm.model_name + ".log";
		SAY_LOG("session " + sessionID + "\n");
		// -------------------------------------------
		// ------------ read parameters --------------
		// -------------------------------------------
		if(CHECK(prm.load_from_file(prm.prm_filename), prm.prm_filename)) return 1;

		// -----------------------------------------------
		// ------------ create test surface --------------
		// -----------------------------------------------
		Surface srf;
		if(CHECK(srf.load_from_file(prm.tet_filename, prm.material_filename), prm.tet_filename + " or " + prm.material_filename)) return 1;

		// ---------------------------------------------
		// ------------ create test rays ---------------
		// ---------------------------------------------
		Ray ray(make_double3(0,0,0), normalize(make_double3(1,0,2)) * (srf.materials[1].Cp), PRayType);
		// Ray(const double3 _r, const double3 _v, const int _type = BaseRayType, const double _A = 1, const double _t = 0):

		// ----------------------------------------------
		// ------------- conduct test computation -------
		// ----------------------------------------------
		//prm.print(cout);
		//srf.print(cout);

		ray.move(&srf, &prm);
	} else {
		cout << "Usage:\n./comp_geo      model_name\n";
	}

	return 0;
}

