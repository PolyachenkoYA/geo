#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : comp_geo.cu
 Author      : PolyachYA
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include "../../geo.cu"

int main(int argc, char **argv)
{

	if(argc >= 2){
		time_t now = time(0);
		string sessionID = string(ctime(&now));
		SAY_LOG("session " + sessionID + "\n");
		// ------------ read params --------------
		string params_filename = string(argv[1]);
		// prm is global. It's not good, but I don't see why.
		// So at least for now it's done this way in case that I don't know how to do better
		if(CHECK(prm.load_from_file(params_filename), params_filename)) return 1;
		if(argc > 2){
			prm.msh_filename = string(argv[2]);
		}

		// ------------ create test surface --------------
		Surface srf;
		if(CHECK(srf.load_from_file(prm.msh_filename), prm.msh_filename)) return 1;

		// ------------ create test rays ---------------
		//Wavefront waves;
		Ray ray(make_double3(0,0,0), normalize(make_double3(1,2,0)));
		//waves.add(ray);

		// ------------- conduct test computation -------
		ray.move(srf);
	} else {
		cout << "Usage:\n./comp_geo      params_file      [msh_file]\n";
	}

	return 0;
}

