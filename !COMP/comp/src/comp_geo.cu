#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : comp_geo.cu
 Author      : PolyachYA
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include "../../geo.cuh"

int main(int argc, char **argv)
{
	if(argc >= 2){
		// --------------------------------------------
		// ------------ handle input ------------------
		// --------------------------------------------
		time_t now = time(0);
		string sessionID = string(ctime(&now));
		Params prm;
		prm.model_name = string(argv[1]);
		prm.prm_filename = prm.model_name + ".prm";
		prm.tet_filename = prm.model_name + ".tet";
		prm.material_filename = prm.model_name + ".mat";
		prm.res_filename = prm.model_name + ".res";
		global_logFname = prm.model_name + ".log";
		SAY_LOG("session " + sessionID + "\n");
		// -------------------------------------------
		// ------------ read parameters --------------
		// -------------------------------------------
		if(CHECK(prm.load_from_file(prm.prm_filename))) return 1;

		// -----------------------------------------------
		// ---------------- read surface -----------------
		// -----------------------------------------------
		Surface srf;
		if(CHECK(srf.load_from_file(prm.tet_filename, prm.material_filename, &prm))) return 1;

		// ----------------------------------------------------
		// --------------- conduct computation ----------------
		// ----------------------------------------------------
		RaysFront rays;
		if(CHECK(compute(&srf, &prm, &rays))) return 1;

		// --------------------------------------------
		// ------------ save results ------------------
		// --------------------------------------------
		string path = "./" + prm.model_name + "/frames/";
	    mkdir(prm.model_name.c_str(),S_IRWXU | S_IRWXG);
	    mkdir(path.c_str(),S_IRWXU | S_IRWXG);

		if(CHECK(prm.save_to_file("./" + prm.model_name + "/" + prm.prm_filename))) return 1;
		if(prm.use_det){
			if(CHECK(srf.saveDetectorInfo(prm.res_filename, &prm))) return 1;
		}
		if(prm.draw_mov){
			if(CHECK(srf.saveMovie(&prm))) return 1;
		}
	} else {
		cout << "Usage:\n./comp_geo      model_name\n";
	}

	return 0;
}

