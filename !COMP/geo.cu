#include "hip/hip_runtime.h"

/**
 * PolyachYA Corporation.  All rights reserved.
 *
 * Please refer to the PolyachYA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include "geo.cuh"

// --------------------------------------------------------------------------------------------
// ------------------------------------ Material ----------------------------------------------
// --------------------------------------------------------------------------------------------

bool Material::isEq(const Material* m2, const double _eps) const
{
	return m2 ? (almostEq(this->Cp, m2->Cp, _eps) && almostEq(this->Cs, m2->Cs, _eps)) : 0;
}

string Material::toStr(string spr1, string spr2)
{
	return spr1 + "Cp = " + toString(this->Cp) + "; Cs = " + toString(this->Cs) + spr2;
}

void Material::print(ostream &output, string spr1, string spr2)
{
	output << this->toStr(spr1, spr2);
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Triangle ----------------------------------------------
// --------------------------------------------------------------------------------------------
Triangle::Triangle(){
	this->clear_params();
}
Triangle::Triangle(const double3 r1, const double3 r2, const double3 r3){
	this->clear_params();
    this->r[0] = r1;
    this->r[1] = r2;
    this->r[2] = r3;
    this->n = this->getNorm();
}
Triangle::Triangle(const double3* r_new)
{
	this->clear_params();
	int i;
	for(i = 0; i < 3; ++i){
		this->r[i] = r_new[i];
	}
	this->n = this->getNorm();
}
Triangle::Triangle(const double3 *vertex, const int* ind)
{
	this->clear_params();
	int i;
	for(i = 0; i < 3; ++i){
		this->i_vrtx[i] = ind[i];
	}
	for(i = 0; i < 3; ++i){
		this->r[i] = vertex[this->i_vrtx[i]];
	}
	this->n = this->getNorm();
}
Triangle::Triangle(const double3 *vertex, const int i0, const int i1, const int i2)
{
	this->clear_params();
	this->i_vrtx[0] = i0;
	this->i_vrtx[1] = i1;
	this->i_vrtx[2] = i2;

	for(int i = 0; i < 3; ++i){
		this->r[i] = vertex[this->i_vrtx[i]];
	}
	this->n = this->getNorm();
}

void Triangle::clear_params()
{
	this->mat[0] = this->mat[1] = nullptr;
	this->detector = nullptr;
	this->is_absorber = 0;
	this->i_vrtx[0] = this->i_vrtx[1] = this->i_vrtx[2] = 0;
	this->r[0] = this->r[1] = this->r[2] = this->n = make_double3(0,0,0);
}

double3 Triangle::getNorm(void) const
{
    return normalize(cross(this->r[1] - this->r[0], this->r[2] - this->r[0]));
}

int Triangle::sg(const double3 rx, const double _eps) const
{
    return sgn(dot(rx, this->n), _eps);
}

int Triangle::isInside(const double3 rx, const double _eps) const
{
    int sg0 = this->sg(cross(this->r[1] - rx, this->r[1] - this->r[0]), _eps);
    return sg0 == 0 ? 0 : ((sg0 == this->sg(cross(this->r[2] - rx, this->r[2] - this->r[1]), _eps)) &&
    		              (sg0 == this->sg(cross(this->r[0] - rx, this->r[0] - this->r[2]), _eps)));
}

string Triangle::ToStr(string spr1, string spr2)
{
	return spr1 + "vertices coords are:\n" +
		   toStr(this->r[0]) + "\n" +
		   toStr(this->r[1]) + "\n" +
		   toStr(this->r[2]) + "\n" +
		   "i_vrtx = " + toString(this->i_vrtx[0]+1) + ";" + toString(this->i_vrtx[1]+1) + ";" + toString(this->i_vrtx[2]+1) + "\n" +
		   "n = " + toStr(this->n) + "\n" +
		   "mat0 : " + this->mat[0]->toStr() + "\n" +
		   "mat1 : " + this->mat[1]->toStr() +  "\n" +
		   "absorber: " + (this->is_absorber ? "yes" : "no") + "\n" +
		   "detector: " + (this->detector ? "yes" : "no") + spr2;
}

void Triangle::print(ostream &output, string spr1, string spr2)
{
	output << this->ToStr(spr1, spr2);
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Ray ---------------------------------------------------
// --------------------------------------------------------------------------------------------

Ray::Ray(Ray *_r)
{
	this->type = _r->type;
	this->c = _r->c;
	this->A = _r->A;
	this->t = _r->t;
	this->r = _r->r;
	this->v = _r->v;
	this->polar = _r->polar;
	this->next = _r->next;
}
Ray::Ray(const double3 _r, const double3 _v, const double3 _polar, const int _type, const double _A, const double _t, Ray *_next):
        type(_type), A(_A), t(_t), next(_next)
{
	this->r = _r;
    this->v = _v;
    this->polar = _polar;
    this->c = length(this->v);
}

RegisteredRay Ray::toRegRay(void)
{
	return RegisteredRay(this);
}

void Ray::add(Ray *ray2)
{
	this->v = (this->v*this->A + ray2->v*ray2->A) / (this->A + ray2->A);
	this->A += ray2->A;
	this->c = ray2->c;
	this->type = ray2->type;
	this->polar = ray2->polar;
	// t & r were determined before
}

int Ray::move(Surface* srf, Params *prm, RaysFront *rays)
{
	if((this->t > prm->Tmax*(1 + prm->eps)) || (this->A < prm->Amin)){
		rays->quit_ray();
		return 0;
	}

	int i;

	// --------------------- pre-geom - find collision point -------------------------
	double3 n, rx;
	pair<double, int> coll_res = this->find_collision(srf, prm);
	// pair<double, int> Ray::find_collision(Surface* srf, Params* prm)
	double dlt_t = coll_res.first;
	int i_coll = coll_res.second;
	Triangle *trng;

	if(i_coll == -1){ // no collision found, so the ray just runs away from the surface
		rays->quit_ray();
		return 0;
	} else {
		trng = &(srf->polygons[i_coll]); // collision surface found
		double3 dlt_r = this->v*dlt_t;

		if(prm->draw_mov){ // draw frames
			double t_new = this->t + dlt_t;
			i = int(this->t / prm->dt) + 1;
			double frame_t = i * prm->dt;
			int3 Xi;
			int ind;
			double3 r_curr;

			// recreate positions of the ray between collisions
			while((frame_t < t_new) && (i < prm->Nfrm)){
				r_curr = this->r + dlt_r * ((frame_t - this->t) / dlt_t); // r when t = frame_t
				// prevent rays from runnig out of the whole system in case of
				// __________|_______|____________________|_____
				//          t_i    t_new                 t_i+1
				if(r_curr.x >= prm->Xmax.x) r_curr.x = prm->Xmax.x - prm->eps;
				if(r_curr.y >= prm->Xmax.y) r_curr.y = prm->Xmax.y - prm->eps;
				if(r_curr.z >= prm->Xmax.z) r_curr.z = prm->Xmax.z - prm->eps;
				if(r_curr.x <= prm->Xmin.x) r_curr.x = prm->Xmin.x + prm->eps;
				if(r_curr.y <= prm->Xmin.y) r_curr.y = prm->Xmin.y + prm->eps;
				if(r_curr.z <= prm->Xmin.z) r_curr.z = prm->Xmin.z + prm->eps;

				switch(prm->prnt_mode){
				case RAW_DATA_MODE:
					srf->frames[i].regRays.push_back(Ray(r_curr, this->v, this->polar, this->type, this->A, frame_t));
					// Ray(const double3 _r, const double3 _v, const int _type = BaseRayType, const double _A = 1, const double _t = 0)
					break;
				case FRAMES_DATA_MODE:
					Xi = get_Nslc(r_curr - prm->Xmin, prm->dX);
					if((Xi.x >= prm->Nslc.x) || (Xi.y >= prm->Nslc.y) || (Xi.z >= prm->Nslc.z)){
						error_handl_string = "r_curr = " + toStr(r_curr) + "; Xi = " + toStr(Xi) + "; Nslc = " + toStr(prm->Nslc) + "; t = " + toString(frame_t);
						return ERROR_MSG;
					}
					ind = ind3D_to_ind(Xi, prm->Nslc);
					if(ind >= srf->frames[i].regRays.size()){
						error_handl_string = "ind = " + toString(ind) + "; rays.size = " + toString(srf->frames[i].regRays.size()) + "; Nslc = " + toStr(prm->Nslc);
						return ERROR_MSG;
					}
					srf->frames[i].regRays[ind].add(this);
					break;
				}
				frame_t += prm->dt;
				++i;
			}
		}

		this->t += dlt_t;
		if(prm->use_det){
			if(trng->detector){ // found triangle is a part of a detector
				trng->detector->regRays.push_back(RegisteredRay(this));
			} // TODO frames - building registered curve on the fly
			if(trng->is_absorber){
				rays->quit_ray();
				return 0;
			}
		}

		rx = this->r + dlt_r; // collision point found
		n = trng->n;
	}

	// phi - angle of incidence
	double cos_phi = fabs(dot(n,this->v)/(length(n)*length(this->v)));

	// ------------------------------ phys angles & amplitude ---------------------------
	double sin_phi = cos_sin(cos_phi);
	double p = sin_phi / this->c;
	//double p2 = p*p;
	double sin_p, sin_s;

	// snell's law
	// TODO sin > 1
	Material *mat_from = trng->mat[ bool2int(trng->sg(trng->r[0] - this->r, 0) < 0) ];
	sin_p = p * mat_from->Cp;
	sin_s = p * mat_from->Cs;

	// --------------------------------- post-geom - create new rays--------------------------------------
	double3 nr = normalize(cross(n,this->v)); // reflection surface
	double3 v_new, r_new;
	double A_new, t_new = this->t + prm->eps;

	double sh_abs = 0, sv_abs = 1;
	if(this->type == SRayType){
		sh_abs = dot(this->polar, nr);
		sv_abs = cos_sin(sh_abs);
	}

	// ---- build new P-ray -------
	A_new = this->A * sv_abs;
	if(sin_p <= 1){
		double cos_p = cos_sin(sin_p);
		v_new = newV(n, this->v, n*cos_p, cross(n, nr)*sin_p) * mat_from->Cp;
		if(is0(v_new)){
			error_handl_string = "v_new for P ray wasn't found for ray " + this->ToStr();
			return ERROR_MSG;
		}

		r_new = rx + v_new * prm->eps;

		rays->add_ray(new Ray(r_new, v_new, normalize(v_new), PRayType, A_new / sqrt(2), t_new));
	} else {
		rays->lostP += A_new * A_new;
	}
	// Ray(const double3 _r, const double3 _v, const int _type = BaseRayType, const double _A = 1, const double _t = 0):
	// newV(double3 n, double3 v, double3 vx, double3 vy)

	// ---- build new S-rays -------
	A_new = this->A;
	if(sin_s <= 1){
		double cos_s = cos_sin(sin_s);
		v_new = newV(n, this->v, n*cos_s, cross(n, nr)*sin_s) * mat_from->Cs;
		if(is0(v_new)){
			error_handl_string = "v_new for S ray wasn't found for ray " + this->ToStr();
			return ERROR_MSG;
		}

		if(sin_p > 1) // r_new wasn't assigned before
			r_new = rx + v_new * prm->eps;

		rays->add_ray(this->type == PRayType ?
					  // this->type == PRayType
					  new Ray(r_new, v_new, normalize(cross(v_new, nr)), SRayType, A_new, t_new) :
					  // this->type == SRayType
					  new Ray(r_new, v_new, normalize((this->polar + nr * sh_abs*(sqrt(2) - 1)) / sqrt(2)), SRayType,
							  A_new*sqrt(sh_abs*sh_abs + sv_abs*sv_abs/2), t_new));
		              // it can be done my creating 2 rays with || and _|_ polarisations, but it's equivalent to a single ray with a rotated polarisation);
	} else {
		rays->lostS += A_new * A_new;
	}

	rays->quit_ray();
	return 0;
}

pair<double, int> Ray::find_collision(Surface* srf, Params* prm)
{
	int i;
	double t, t_min = prm->Tmax * 2;
	Triangle *trngl;
	int i_coll = -1; // indicator of no found collision

	for(i = 0; i < srf->Npol; ++i){ // find collision point

		trngl = &(srf->polygons[i]); // so we don't have to call [i] every time. Also it's shorter
		t = dot(trngl->r[0] - this->r, trngl->n) / dot(this->v, trngl->n); // find time of collision

		if((SYS_EPS < t) && (t < t_min)){
		// if the possible collision can happen (t > 0) and if it's better than the one we already have (t < t_min)
			if(trngl->isInside(this->r + this->v*t, prm->eps)){ // if it's really the point, then save it
				t_min = t;
				i_coll = i;
			}
		}
	}

	return make_pair(t_min, i_coll);
}

string Ray::ToStr(const string spr)
{
	return spr + (this->type == PRayType ? "P-type" : "S-type") +
			"\nc = " + toString(this->c) + "; A = " + toString(this->A) + "; t = " + toString(this->t) +
			"\nr = " + toStr(this->r) +
			"\nv = " + toStr(this->v) + spr;
}

void Ray::print(ostream &output, string spr)
{
	output << this->ToStr(spr);
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ RayFront ----------------------------------------------
// --------------------------------------------------------------------------------------------

void RaysFront::add_ray(Ray *new_ray)
{
	this->last_ray->next = new_ray;
	this->last_ray = this->last_ray->next;
	this->inc_rays();
}

void RaysFront::shift_current_ray(void)
{
	Ray *old_ray = this->current_ray;
	this->current_ray = this->current_ray->next;
	delete old_ray;
}

void RaysFront::quit_ray(void)
{
	--this->n_alive_rays;
	this->shift_current_ray();
}

void RaysFront::inc_rays(void)
{
	++this->n_alive_rays;
	++this->n_total_rays;
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Surface -----------------------------------------------
// --------------------------------------------------------------------------------------------

Surface::Surface(int _n_pol, int _n_mat, int _n_det)
{
	this->resize_clear(_n_pol, _n_mat, _n_det);
}

void Surface::resize_clear(int _n_pol, int _n_mat, int _n_det)
{
	if(_n_pol != this->Npol){
		delete[] this->polygons;
		this->polygons = (_n_pol == 0 ? nullptr : (new Triangle[_n_pol]));
		this->Npol = _n_pol;
	}
	if(_n_mat != this->Nmat){
		delete[] this->materials;
		this->materials = (_n_mat == 0 ? nullptr : (new Material[_n_mat]));
		this->Nmat = _n_mat;
	}
	if(_n_det != this->Ndet){
		delete[] this->detectors;
		this->detectors = (_n_det == 0 ? nullptr : (new Detector[_n_det]));
		this->Ndet = _n_det;
	}
}

void Surface::print(ostream &output, string spr1, string spr2)
{
	int i;
	output << spr1 << "---------- Materials ----------\n"
		   << "Nmat = " << this->Nmat << "\n";
	for(i = 0; i < this->Nmat; ++i){
		this->materials[i].print(output, toString(i+1) + ") ");
	}
	output << "---------- Polygons ----------\n"
		   << "Npol = " << this->Npol << "\n";
	for(i = 0; i < this->Npol; ++i){
		this->polygons[i].print(output, "\n------------\n" + toString(i+1) + "\n");
	}
	output << spr2;
}

int Surface::load_from_file(string surf_filename, string mat_filename, Params *prm)
{
	ifstream input;
	int i, i2, i3, i4, i5;
	string buf_str;

	// --------------------------------------- read materials -----------------------------------------------
	input.open(mat_filename);
	if(!input){
		input.close();
		error_handl_string = "material-file " + mat_filename + " is missing\n";
		return CANT_OPEN_FILE_FOR_READING;
	}

	int _n_mat;
	input >> _n_mat;
	if(_n_mat < 2){
		input.close();
		error_handl_string = "Nmat = " + toString(_n_mat) + "; must be at least 2\n";
		return LESS_2_MATERIALS;
	}
	this->resize_clear(this->Npol, _n_mat, this->Ndet);
	for(i = 0; i < this->Nmat; ++i){
		input >> this->materials[i].Cp >> this->materials[i].Cs;
	}
	input.close();


	// ---------------------------------------------------------------------------------------------------
	// ------------------------------------- read surface ------------------------------------------------
	// ---------------------------------------------------------------------------------------------------
	input.open(surf_filename);
	if(!input){
		input.close();
		error_handl_string = "surface-file " + surf_filename + " is missing\n";
		return CANT_OPEN_FILE_FOR_READING;
	}

	// --------------------------------------- read vertices -----------------------------------------------
	double3 *vertex;
	int Nvertex;

	input >> Nvertex;
	vertex = new double3[Nvertex];
	bool find_Xbounds = areEq(prm->Xmax, prm->Xmin);
	if(find_Xbounds){
		prm->Xmax = make_double3(-9999999999999.0, -9999999999999.0, -9999999999999.0);
		prm->Xmin = make_double3(9999999999999.0, 9999999999999.0, 9999999999999.0);
	}

	for(i = 0; i < Nvertex; ++i){
		input >> vertex[i].x >> vertex[i].y >> vertex[i].z;

		if(find_Xbounds){ // find global borders
			if(vertex[i].x > prm->Xmax.x) prm->Xmax.x = vertex[i].x;
			if(vertex[i].x < prm->Xmin.x) prm->Xmin.x = vertex[i].x;
			if(vertex[i].y > prm->Xmax.y) prm->Xmax.y = vertex[i].y;
			if(vertex[i].y < prm->Xmin.y) prm->Xmin.y = vertex[i].y;
			if(vertex[i].z > prm->Xmax.z) prm->Xmax.z = vertex[i].z;
			if(vertex[i].z < prm->Xmin.z) prm->Xmin.z = vertex[i].z;
		}
	}

	// now we know real Xmax & Xmin & dX
	// so we can allocate memory for frames (and for the grid if necessary)
	if(prm->dX.x == 0) prm->dX.x = (prm->Xmax.x - prm->Xmin.x)*1.01; // bigger than max delta, so int(delta/dX) == 0 ,
	if(prm->dX.y == 0) prm->dX.y = (prm->Xmax.y - prm->Xmin.y)*1.01; // so Nslc = 1 in the end
	if(prm->dX.z == 0) prm->dX.z = (prm->Xmax.z - prm->Xmin.z)*1.01;
	prm->Nslc = get_Nslc(prm->Xmax - prm->Xmin, prm->dX) + 1;

	if(prm->draw_mov){
		this->frames = new Frame[prm->Nfrm + 1];
		if(prm->prnt_mode == FRAMES_DATA_MODE){
			int Nnodes = prm->Nslc.x * prm->Nslc.y * prm->Nslc.z;
			int ind;
			int3 Xi;

			for(i = 0; i < prm->Nfrm; ++i){
				this->frames[i].regRays.resize(Nnodes);
				/*
				 * now (27.07.2018) sizeof(Ray)==112, so here we need a lot of RAM
				 * In fact we don't need fields "c","t","next" for registered rays
				 * so we can create a special class for frame-registered rays which would be ~90 bytes.
				 * It's not a big difference, so I didn't bother so far, but it can be done at any moment.
				 */
				for(Xi.z = 0; Xi.z < prm->Nslc.z; ++Xi.z) for(Xi.y = 0; Xi.y < prm->Nslc.y; ++Xi.y) for(Xi.x = 0; Xi.x < prm->Nslc.x; ++Xi.x){
					ind = ind3D_to_ind(Xi, prm->Nslc);
					this->frames[i].regRays[ind].t = i * prm->dt;
					this->frames[i].regRays[ind].r = (Xi + 0.5) * prm->dX + prm->Xmin;
				}
			}
		}
	}

	// ------------------------------------- read tetrahedrons & triangles ---------------------------------------------
	int Ntet, i_mat, i_vertex[4];

	input >> Ntet;

	vector<Triangle> read_polygons;
	int n_overlap_vrtx, i_curr_mat;
	int tet_surf_ind[4][3] =
	// tet_surf_ind[i][:] - all vertices except i-th one -
	// the indices of the i-th triangle-sub-surf of the abstract tetrahedron
	{
	  {1, 2, 3},
	  {0, 2, 3},
	  {0, 1, 3},
	  {0, 1, 2}
	};

	for(i = 0; i < Ntet; ++i){
		for(i2 = 0; i2 < 4; ++i2){ // read vertices of tetrahedron_i
			input >> i_vertex[i2];
			--i_vertex[i2]; // arrays from 0, in file-format from 1
			if(i_vertex[i2] >= Nvertex){
				error_handl_string = "index of the " + toString(i2) + "th vertex of the " + toString(i) + "th tetrahedron is " + toString(i_vertex[i2]) + "; Nvertex = " + toString(Nvertex) + "\n";
				delete[] vertex;
				read_polygons.clear();
				input.close();
				return ERROR_MSG;
			 }
			// TODO check for identical vertices in a single tetrahedron
		}
		input >> i_mat; // read material of tetrahedron_i
		// materils are indexed from 0, but 0th material is for background, so we don't do --i_mat
		if(i_mat >= this->Nmat){
			error_handl_string = "index of material of the " + toString(i) + "th tetrahedron is " + toString(i_mat) + "; Nmat = " + toString(this->Nmat) + "\n";
			delete[] vertex;
			read_polygons.clear();
			input.close();
			return ERROR_MSG;
		}

		for(i2 = 0; i2 < 4; ++i2){                        // for each tetrahedron sub-surface (tet_surf_ind[i2][:]) check overlaps
			for(i3 = 0; i3 < read_polygons.size(); ++i3){ // with all existing ones
				n_overlap_vrtx = 0;
				for(i4 = 0; i4 < 3; ++i4){
					for(i5 = 0; i5 < 3; ++i5){
						/*
						 * i - index of the current tetrahedron
						 * i2 - index of surface of current tetrahedron
						 * i3 - for all existing sufraces - index of some existing surface to check a pair tetr_surf<->existing_surf
						 * i4 - index of the vertex of the current surf of the current tetr
						 * i5 - index of the vertex of the current existing triangle
						 */
						if(i_vertex[ tet_surf_ind[i2][i4] ] == read_polygons[i3].i_vrtx[i5]){
							++n_overlap_vrtx;
							break;
						}
					}
				}
				if(n_overlap_vrtx >= 3){
					break;
					/*
					 * current (i2) triangle-sub-surf can't overlap with more than 1 existing triangle.
					 * It's found, so no overlapping with all further ones.
					 * */
				}
			}
			if(n_overlap_vrtx > 3){
				error_handl_string = "n_overlap_vrtx = " + toString(n_overlap_vrtx) + "\n i = " + toString(i) + "; i2 = " + toString(i2) + "; i3 = " + toString(i3);
				delete[] vertex;
				read_polygons.clear();
				input.close();
				return ERROR_MSG;
			}
			if(i3 == read_polygons.size()){ // current (i2) triangle-sub-surf is new to the all existing ones
				read_polygons.push_back(Triangle(vertex, i_vertex[ tet_surf_ind[i2][0] ],
						                                 i_vertex[ tet_surf_ind[i2][1] ],
						                                 i_vertex[ tet_surf_ind[i2][2] ]));
			}
			// else means already used polygons similar to the current (i2) one was found
			// read_polygons[i3].i_vrtx[:] == i_vertex[ tet_surf_ind[i2][:] ]

			/*
			 * if the current triangle is a new one, the i3 is the number of polygons BEFORE new one was added,
			 * so after all i3 is the index of the last added triangle. In this case i3 == read_polygons.size()-1
			 *
			 * otherwise (existing similar one was found) i3 is the index of the triangle identical to the current one.
			 * In this case we add 2nd matetrial to the existing polygon and don't add a new triangle to polygons
			 *
			 * So in both cases i3 is the index of the triangle to add the current material to
			 */
			i_curr_mat = bool2int(read_polygons[i3].sg(vertex[ i_vertex[i2] ] - read_polygons[i3].r[0], 0) > 0);
			// i2 is the missing index in tet_surf_ind[i2][:] set, so i_vertex[i2] is the 4th vertex of the current tetrahedron
			read_polygons[i3].mat[i_curr_mat] = this->materials + i_mat;

			if(read_polygons[i3].mat[i_curr_mat]->isEq(read_polygons[i3].mat[bool2int(!i_curr_mat)])){
				// read_polygons[i3] is a formal border with no real meaning
				read_polygons.erase(read_polygons.begin()+i3);
			}
		}
	}
	/*
	 * As a result we have read_polygons with all true-surface triangles of the system
	 * Now we need to add background material and copy read polygons to the main array
	 */
	for(i = 0; i < read_polygons.size(); ++i){ // if some materials are unset, then set them to background material
		for(i2 = 0; i2 < 2; ++i2){
			if(!read_polygons[i].mat[i2]){
				read_polygons[i].mat[i2] = this->materials; // &(material[0])
			}
		}
		if(read_polygons[i].mat[0]->isEq(read_polygons[i].mat[1])){
			// read_polygons[i] is a formal border with no real meaning
			// actually it can't happen here if everything before worked fine.
			read_polygons.erase(read_polygons.begin()+i);

			error_handl_string = toString(i) + "-th polygon has no material set\n";
			delete[] vertex;
			read_polygons.clear();
			input.close();
			return ERROR_MSG;
		}
	}

	this->resize_clear(read_polygons.size(), this->Nmat, this->Ndet);
	for(i = 0; i < this->Npol; ++i){ // copy read data
		this->polygons[i] = read_polygons[i];
	}
	read_polygons.clear();
	delete[] vertex;

	// --------------------------------------- read detectors -----------------------------------------------
	if(prm->use_det){
		int Ntrg, i_det, _n_det;
		Triangle *trng;

		input >> Ntrg >> _n_det;
		this->resize_clear(this->Npol, this->Nmat, _n_det);
		for(i = 0; i < Ntrg; ++i){
			for(i2 = 0; i2 < 3; ++i2){
				input >> i_vertex[i2];
				--i_vertex[i2];
				if(i_vertex[i2] >= Nvertex){
					error_handl_string = "index of the " + toString(i2) + "-th vertex of the " + toString(i) + "-th tetrahedron is " + toString(i_vertex[i2]) + "; Nvertex = " + toString(Nvertex) + "\n";
					input.close();
					return ERROR_MSG;
				 }
			}
			input >> i_det;
			--i_det;
			// --i is necessary because all arrays are indexed from 0 but all abjects in the file are indexed from 1
			if(i_det >= this->Ndet){
				error_handl_string = "index of the detector of the " + toString(i) + "-th polygon is " + toString(i_det) + "; Ndet = " + toString(this->Ndet) + "\n";
				input.close();
				return ERROR_MSG;
			}

			trng = this->findPolygon(i_vertex);
			if(trng){
				if(i_det >= 0){
					trng->detector = &(this->detectors[i_det]); // add i2-th polygon to the i_det-th detector
				}
				trng->is_absorber = 1; // just an absorbing polygon, not a detector
			} else {
				error_handl_string = toString(i) + "-th polygon is missing in the final version of the surface\n";
				input.close();
				return ERROR_MSG;
			}
		}
	}

	input.close();

	return 0;
}

Triangle* Surface::findPolygon(int* i_vertex) // find triangle by its vertices
{
	int i, i2, i3;
	int n_overlap_vrtx;

	for(i = 0; i < this->Npol; ++i){     // for each polygon
		n_overlap_vrtx = 0;
		for(i2 = 0; i2 < 3; ++i2){          // i_vertex[i2]
			for(i3 = 0; i3 < 3; ++i3){      // this->polygons[i].i_vrtx[i3]
				if(i_vertex[i2] == this->polygons[i].i_vrtx[i3]){
					++n_overlap_vrtx; // count overlapping vertices
					//cout << i2 << " " << i << " " << i3 << "\n";
				}
			}
		}
		if(n_overlap_vrtx >= 3){  // i - found polygon index
			if(n_overlap_vrtx > 3){
				error_handl_string = "Polygon (" + toString(i_vertex[0]) + ";" + toString(i_vertex[0]) + ";" + toString(i_vertex[0]) + ") has too many overlaps with " + toString(i) + "th polygon";
				CHECK(ERROR_MSG);
				exit(1);
			}
			break;
		}
	}

	// i == this->Npol means no existing polygon matches all 3 vertices
	return (i == this->Npol ? nullptr :  &(this->polygons[i]));
}

int Surface::saveDetectorInfo(string filename, Params *prm)
{
	ofstream output;

	output.open(filename);
	if(!output){
		output.close();
		error_handl_string = "can not open output file " + filename + " for writing\n";
		return CANT_OPEN_FILE_FOR_WRITING;
	}

	int i;
	// prm->Nfrm = (int)(prm->Tmax*(1 + prm->eps) / prm->dt)+1;
	output << this->Ndet << " " << prm->Nfrm << "\n";
	for(i = 0; i < this->Ndet; ++i){
		//output << i+1 << "\n";
		this->detectors[i].saveInfo(output, prm);
	}

	output.close();
	return 0;
}

int Surface::saveMovie(Params *prm)
{
	string path = "./" + prm->model_name + "/frames/";
    int i;
	ofstream output;
	string filename;

	time_t real_start_t;

	double tot_rays = 0, rays_printed = 0;
	for(i = 0; i < prm->Nfrm; ++i){
		tot_rays += this->frames[i].regRays.size();
	}

	time(&real_start_t);
	for(i = 0; i < prm->Nfrm; ++i){
		filename = path + toString(i) + ".frm";
		output.open(filename);
		if(!output){
			output.close();
			error_handl_string = "can not open output file \n|" + filename + "|\nfor writing\n";
			return CANT_OPEN_FILE_FOR_WRITING;
		}

		this->frames[i].saveToFile(output, prm);
		rays_printed += this->frames[i].regRays.size();

		output.close();

		time_progress(real_start_t, time(0), rays_printed / tot_rays, "saving movie");
	}

	return 0;
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Frame -------------------------------------------------
// --------------------------------------------------------------------------------------------

int Frame::saveToFile(ostream &output, Params *prm)
{
	int i;

	switch(prm->prnt_mode){ // TODO PRNT_MODE_ID
	case RAW_DATA_MODE:
	case FRAMES_DATA_MODE:
		int real_N = 0;
		for(i = 0; i < this->regRays.size(); ++i){
			if(this->regRays[i].A > 0)
				++real_N;
		}

		Ray *curr_ray;
		output << real_N << "\n";
		for(i = 0; i < this->regRays.size(); ++i){
			curr_ray = &(this->regRays[i]);
			if(curr_ray->A > 0){
				output //<< i << " "
				   	   << (curr_ray->type - BaseRayType) << " "
				   	   << curr_ray->t << " "
				   	   << curr_ray->A << " "
				   	   << curr_ray->c << " "
				   	   << toStr(curr_ray->r, "", " ", "") << " "
				   	   << toStr(curr_ray->v, "", " ", "") << "\n";
			}
		}
		break;
	}

	return 0;
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Detector ----------------------------------------------
// --------------------------------------------------------------------------------------------

void Detector::saveInfo(ostream &output, Params *prm)
{
	int i;
	switch(prm->prnt_mode){
		case RAW_DATA_MODE:
			// save raw data
			for(i = 0; i < this->regRays.size(); ++i){
				output << i << " " << this->regRays[i].t << " " << this->regRays[i].A << " " << this->regRays[i].c << " " << this->regRays[i].type-RayTimeIND << "\n";
			}
			break;
		case FRAMES_DATA_MODE:
			// save registered curve
			double *regArr = new double[prm->Nfrm];

#ifdef _OPENMP
#pragma omp parallel for
#endif
			for(i = 0; i < prm->Nfrm; ++i){
				regArr[i] = this->regValue(prm->dt*i, prm);
			}

			for(i = 0; i < prm->Nfrm; ++i){
				output << prm->dt*i << " " << regArr[i] << "\n";
			}

			delete[] regArr;
	}
}

double Detector::regValue(double t0, Params *prm)
{
	int i;
	double res = 0;

	for(i = 0; i < this->regRays.size(); ++i){
		// res += this->peakFnc((t0 - this->regRays[i].t) / prm->tau, prm->B, this->regRays[i].A);
		res += this->peakFnc((t0 - this->regRays[i].t) / prm->tau, prm->B);
	}

	return res / prm->Nrays;
}

double Detector::peakFnc(double x, double B, double A)
// y = A*cos(2pi*f*t)*cos^4(t/tau) =
// = A*cos(B*x)*cos^4(x), x = (t-t0)/tau, B = f*tau
{
	x = std::abs(x);
	if(x > pi_d2){
		return 0;
	} else {
		double _b = float_part(B*x);
		if((0.25 <= _b) && (_b <= 0.75)) // cos(B*x) < 0
			return 0;
		_b = cos(x);
		_b = _b * _b; // cos^2
		_b = A * cos(pi_m2 * B * x) * _b * _b;
		return  _b;
	}
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Params ------------------------------------------------
// --------------------------------------------------------------------------------------------

int Params::load_from_file(string filename)
{
	ifstream input(filename);
	if(!input){
		input.close();
		error_handl_string = "file " + filename + " is missing\n";
		return ERROR_MSG;
	}

	string buf_s;
	std::getline(input, buf_s); // read comment line
	input >> this->Nrays >> this->Tmax >> this->dt >> this->Amin >> this->f >> this->tau >> this->eps
	      >> this->use_det >> this->draw_mov >> this->prnt_mode
	      >> this->Xmin.x >> this->Xmin.y >> this->Xmin.z
	      >> this->Xmax.x >> this->Xmax.y >> this->Xmax.z
	      >> this->dX.x >> this->dX.y >> this->dX.z;
	this->prnt_mode += PRINT_MODE_ID;

	input.close();

	if(this->eps == 0) this->eps = SYS_EPS;
	if(this->dt > 0){
		this->Nfrm = (int)(this->Tmax*(1 + this->eps) / this->dt)+1;
	} else {
		this->Nfrm = 0;
		if((this->prnt_mode == FRAMES_DATA_MODE) && this->use_det){
			error_handl_string = "average-sum result for detectors is requested, but dt <= 0\n";
			return ERROR_MSG;
		}
		if(draw_mov){
			error_handl_string = "draw movie requested, but dt <= 0\n";
			return ERROR_MSG;
		}
	}

	this->B = this->f * this->tau;
	this->f *= TIME_UNIT;   //  Gz * 10^6
	this->tau /= TIME_UNIT; // sec * 10^-6

	return 0;
}

void Params::print_full(ostream &output, string spr)
{
	for(int i = 0; i < this->paramFHead.size(); ++i){
		output << this->paramFHead[i] << " | ";
	}
	output << spr << this->Nrays << spr
		   << this->Tmax << spr
		   << this->Amin << spr
		   << this->eps << spr
		   << "model_name: |" << this->model_name << "|" << spr;
		   //<< "alive_rays: " << this->n_alive_rays << "; total_rays: " << this->n_total_rays << spr;
}

void Params::print(ostream &output)
{
	int spForVal = 15;

	for(int i = 0; i < this->paramFHead.size(); ++i){
		output << setw(spForVal) << this->paramFHead[i];
	}

	output << "\n"
		   << setw(spForVal) << this->Nrays << setw(spForVal) << this->Tmax << setw(spForVal) << this->dt
		   << setw(spForVal) << this->Amin << setw(spForVal) << this->f/TIME_UNIT << setw(spForVal) << this->tau*TIME_UNIT
		   << setw(spForVal) << this->eps << setw(spForVal) << this->use_det << setw(spForVal) << this->draw_mov
		   << setw(spForVal) << (this->prnt_mode - PRINT_MODE_ID)
		   << setw(spForVal) << this->Xmin.x << setw(spForVal) << this->Xmin.y << setw(spForVal) << this->Xmin.z
		   << setw(spForVal) << this->Xmax.x << setw(spForVal) << this->Xmax.y << setw(spForVal) << this->Xmax.z
		   << setw(spForVal) << this->dX.x << setw(spForVal) << this->dX.y << setw(spForVal) << this->dX.z;
}

int Params::save_to_file(string filename)
{
	ofstream output(filename);
	if(!output){
		error_handl_string = "can not open file                                    \n|" +
				              filename +
				              "|                                             \nfor writing                                           \n";
		return CANT_OPEN_FILE_FOR_WRITING;
	}

	this->print(output);

	output.close();
	return 0;
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Global Fncs -------------------------------------------
// --------------------------------------------------------------------------------------------

int compute(Surface *srf, Params *prm, RaysFront *rays)
{
	time_t real_start_t;
	int i;

	double3 *rays_v = new double3[prm->Nrays];
	// generate rays
	for(i = 0; i < prm->Nrays; ++i){
		rays_v[i] = vecByAngles(myRnd(pi/4, 3*pi/4), 0) * srf->materials[1].Cp; // v || oY
		//rays_v[i] = vecByAngles(myRnd(pi/4, 3*pi/4), myRnd(-pi/5, pi/5)) * srf->materials[1].Cp; // v || oY
	}
	/*
	for(i = 0; i < prm->Nrays/4; ++i){
		// rays_v[4*i] = rndVec(srf->materials[1].Cp);
		rays_v[4*i]   = vecByAngles(myRnd(-pi/4-0.1, pi/4+0.1), myRnd(-pi/4-0.1, pi/4+0.1)) * srf->materials[1].Cp;
		rays_v[4*i+1] = vecByAngles(myRnd(pi/4-0.1, pi*3/4+0.1), myRnd(-pi/4-0.1, pi/4+0.1)) * srf->materials[1].Cp;
		rays_v[4*i+2] = vecByAngles(myRnd(pi*3/4-0.1, pi*5/4+0.1), myRnd(-pi/4-0.1, pi/4+0.1)) * srf->materials[1].Cp;
		rays_v[4*i+3] = vecByAngles(myRnd(pi*5/4-0.1, pi*7/4+0.1), myRnd(-pi/4-0.1, pi/4+0.1)) * srf->materials[1].Cp;
		// vecByAngles(myRnd(-pi, pi), myRnd(-pi_d2, pi_d2))*V
	}
	*/

	time(&real_start_t);
	for(i = 0; i < prm->Nrays; ++i){
		rays->current_ray = new Ray(make_double3(0,0,0), rays_v[i], normalize(rays_v[i]), PRayType, 1, 0);
		// Ray(const double3 _r, const double3 _v, const double3 _polar, const int _type = PRayType, const double _A = 1, const double _t = 0, const Ray *_next = nullptr);
		rays->inc_rays();
		rays->last_ray = rays->current_ray;
		rays->totalE0 += rays->current_ray->A * rays->current_ray->A;

		do{
			if(rays->current_ray->move(srf, prm, rays)){
				error_handl_string += ("\n" + toString(i) + "-th ray failed\n");
				delete[] rays_v;
				return ERROR_MSG;
			}
		}while(rays->current_ray); // while there are rays to compute

		time_progress(real_start_t, time(0), (i+1) / (float(prm->Nrays)), "computing");
	}

	cout << "lostP/totalE = " << rays->lostP / rays->totalE0 << "                                      \n"
		 << "lostS/totalE = " << rays->lostS / rays->totalE0 << "                                      \n";

	delete[] rays_v;
	return 0;
}
