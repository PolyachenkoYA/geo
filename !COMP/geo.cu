#include "hip/hip_runtime.h"

/**
 * PolyachYA Corporation.  All rights reserved.
 *
 * Please refer to the PolyachYA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include "geo.cuh"

// ----------------------------------------- main math ------------------------------------------

int rightV(vector<double3>& v, double3 n, double3 v0)
{
	if(v.size() != 4){
		CHECK(WRONG_SET_OF_POSSIBLE_REFLECTED_RAYS, 0);
		exit(1);
	}


	int i,res = -1;
	double cs, max_cs = -2, kv = 1/length(v0);

	// reflected ray have the least angle with v0 |=> the beggest cos(v_new,v0)
	for(i = 0; i < 4; ++i){
		if(dot(v0, n)*dot(v[i], n) < 0){ // exclude other side of surface
			cs = dot(v[i], v0)/length(v[i])*kv;
			if(cs > max_cs){ // determine reflected ray
				res = i;
				max_cs = cs;
			}
		}
	}

	if(res == -1){
		CHECK(NO_REFLECTED_RAY_FOUND, 0);
		exit(1);
	}
	return res;
}

double3 newV(double3 n, double3 v_old, double3 vx, double3 vy)
{
	vector<double3> v_news;
	v_news.clear();
	// we don't know the direction of n and nr, so we have to check all 4 possible variants
	// to choose the one that really is the physicaly reflected ray
	v_news.push_back(vx + vy);
	v_news.push_back(vx - vy);
	v_news.push_back(-v_news[1]);
	v_news.push_back(-v_news[0]);
	return v_news[rightV(v_news, n, v_old)];
}

// ----------------------------------------- Other stuff ----------------------------------------------
char bool2int(bool b){ return b ? 1 : 0; }

template<typename T>
//int stp(T str)
void stp(T str)
{
	cerr << str << endl;
	cin.get();
	//int r;
	//cin >> r;
	//return r;
}

template<typename T>
string vectorToStr(vector<T> v, string sp)
{
        string s="";
        for(int i = 0; i < v.size()-1; ++i) s+= (toString(v[i])+sp);
        return s+toString(v[v.size()-1]);
}
template<typename T>
void printVector(ostream &output, vector<T> v, string sp1, string sp2, string sp3)
{
	int sz = v.size();
    output << sp1;
    for(int i = 0; i < sz-1; ++i) output << v[i] << sp2;
    output << v[sz-1] << sp3;
}
template<typename T>
T sumVector(vector<T> v)
{
        T s=0;
        for(int i = 0; i < v.size(); ++i) s+=v[i];
        return s;
}
template <typename T>
string toString(T val)
{
    std::ostringstream oss;
    oss << val;
    return oss.str();
}
template<typename T>
T fromString(const string& s)
{
  std::istringstream iss(s);
  T res;
  iss >> res;
  return res;
}

string toLower(string s)
{
	char d = 'a'-'A';
	for(int i = 0; i<s.size(); ++i) if((s[i]>='A') && (s[i]<='Z')) s[i]+=d;
	return s;
}
string toUpper(string s)
{
	char d = 'A'-'a';
	for(int i = 0; i<s.size(); ++i) if((s[i]>='a') && (s[i]<='z')) s[i]+=d;
	return s;
}
void printD3(ostream &output, double3 r, string sp1, string sp2, string sp3)
{
	output << d3ToStr(r, sp1, sp2, sp3);
}

template<typename T>
int CHECK(int n, T s)
{
    if(!n){ return 0; }
    ofstream Fout(global_logFname.c_str(), ios::app);
    if(!Fout){ return CANT_OPEN_FILE_FOR_WRITING; }

    if(n != SAY_IT){
    	string _s = "error #" + toString(n) + "\n   message:\n" + error_handl_string + "\n";
    	Fout << "\n" << _s;
    	cout << _s;
    }
    switch(n){
    	case SAY_IT: Fout << s; break;

    	case CANT_OPEN_FILE_FOR_READING: Fout << "Can't open file\n" << s << "\nfor reading"; break;
    	case CANT_OPEN_FILE_FOR_WRITING: Fout << "Can't open file\n" << s << "\nfor writing"; break;

    	case WRONG_SET_OF_POSSIBLE_REFLECTED_RAYS: Fout << "Wrong set of possible reflected rays (v.size != 4)\n"; break;
    	case NO_REFLECTED_RAY_FOUND: Fout << "No reflected ray found\n"; break;
    	case WRONG_RAY_TYPE: Fout << "Wrong ray type: " << s; break;

        default: Fout << "Unknown error #" << n << "\nerror message:\n" << s;
    }
    if(n != SAY_IT){ Fout << "\n"; }
    Fout.close();

    return n;
}

template<typename T>
void SAY_LOG(T s)
{
	CHECK(SAY_IT, s);
}

// ---------------------------------------- Other math -------------------------------------------------
bool almostEq(double x, double y, double _eps)
{
	return y == 0 ? (abs(x) < _eps) : (abs(x/y - 1) < _eps);
}

string d3ToStr(double3 d, string sp1, string sp2, string sp3)
{
	return sp1 + toString(d.x) + sp2 + toString(d.y) + sp2 + toString(d.z) + sp3;
}

vector<double> d3ToV(double3 v){
	double vp[3] = {v.x, v.y, v.z};
	vector<double> vv;
	vv.assign(vp, vp+3);
	return vv;
}

double cos_sin(double x)
{
    return sqrtf(1-x*x);
}

double sqr(double x)
{
	return x*x;
}

int sgn(double x, double _eps)
{
    if(x >= _eps){
    	return 1;
    } else if(x <= -_eps){
        return -1;
    } else {
        return 0;
    }
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Material ----------------------------------------------
// --------------------------------------------------------------------------------------------

bool Material::isEq(const Material* m2, const double _eps) const
{
	return m2 == NULL ? 0 : (almostEq(this->Cp, m2->Cp, _eps) && almostEq(this->Cs, m2->Cs, _eps));
}

string Material::toStr(string spr1, string spr2)
{
	return spr1 + "Cp = " + toString(this->Cp) + "; Cs = " + toString(this->Cs) + spr2;
}

void Material::print(ostream &output, string spr1, string spr2)
{
	output << this->toStr(spr1, spr2);
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Triangle ----------------------------------------------
// --------------------------------------------------------------------------------------------
Triangle::Triangle(){
	this->clear_params();
}
Triangle::Triangle(const double3 r1, const double3 r2, const double3 r3){
	this->clear_params();
    this->r[0] = r1;
    this->r[1] = r2;
    this->r[2] = r3;
    this->n = this->getNorm();
}
Triangle::Triangle(const double3* r_new)
{
	this->clear_params();
	int i;
	for(i = 0; i < 3; ++i){
		this->r[i] = r_new[i];
	}
	this->n = this->getNorm();
}
Triangle::Triangle(const double3 *vertex, const int* ind)
{
	this->clear_params();
	int i;
	for(i = 0; i < 3; ++i){
		this->i_vrtx[i] = ind[i];
	}
	for(i = 0; i < 3; ++i){
		this->r[i] = vertex[this->i_vrtx[i]];
	}
	this->n = this->getNorm();
}
Triangle::Triangle(const double3 *vertex, const int i0, const int i1, const int i2)
{
	this->clear_params();
	this->i_vrtx[0] = i0;
	this->i_vrtx[1] = i1;
	this->i_vrtx[2] = i2;

	for(int i = 0; i < 3; ++i){
		this->r[i] = vertex[this->i_vrtx[i]];
	}
	this->n = this->getNorm();
}

void Triangle::clear_params()
{
	this->mat[0] = this->mat[1] = NULL;
	this->i_vrtx[0] = this->i_vrtx[1] = this->i_vrtx[2] = 0;
	this->r[0] = this->r[1] = this->r[2] = this->n = make_double3(0,0,0);
}

double3 Triangle::getNorm(void) const
{
    return normalize(cross(this->r[1] - this->r[0], this->r[2] - this->r[0]));
}

int Triangle::sg(const double3 rx, const double _eps) const
{
    return sgn(dot(rx, this->n), _eps);
}

int Triangle::isInside(const double3 rx, const double _eps) const
{
    int sg0 = this->sg(cross(this->r[1] - rx, this->r[1] - this->r[0]), _eps);
    return sg0 == 0 ? 0 : ((sg0 == this->sg(cross(this->r[2] - rx, this->r[2] - this->r[1]), _eps)) &&
    		              (sg0 == this->sg(cross(this->r[0] - rx, this->r[0] - this->r[2]), _eps)));
}

string Triangle::toStr(string spr1, string spr2)
{
	return spr1 + "vertices coords are:\n" +
		   d3ToStr(this->r[0]) + "\n" +
		   d3ToStr(this->r[1]) + "\n" +
		   d3ToStr(this->r[2]) + "\n" +
		   "i_vrtx = " + toString(this->i_vrtx[0]+1) + ";" + toString(this->i_vrtx[1]+1) + ";" + toString(this->i_vrtx[2]+1) + "\n" +
		   "n = " + d3ToStr(this->n) + "\n" +
		   "mat0 : " + this->mat[0]->toStr() + "\n"
		   "mat1 : " + this->mat[1]->toStr() + spr2;
}

void Triangle::print(ostream &output, string spr1, string spr2)
{
	output << this->toStr(spr1, spr2);
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Ray ---------------------------------------------------
// --------------------------------------------------------------------------------------------

void Ray::move(Surface* srf, Params *prm)
{
	this->print(cout);
	prm->n_alive_rays++;
	prm->n_total_rays++;

	if((this->t + length(this->r)/this->c > prm->Tmax) || (this->A < prm->Amin)){
		prm->n_alive_rays--;
		return;
	}

	//int i;

	// --------------------- pre-geom - find collision point -------------------------
	double3 n, rx;
	pair<double, int> coll_res = this->find_collision(srf, prm);
	// pair<double, int> Ray::find_collision(Surface* srf, Params* prm)
	double t0 = coll_res.first;
	int i_coll = coll_res.second;
	Triangle *trng;

	if(i_coll == -1){
		prm->n_alive_rays--;
		return;
	} else {
		rx = this->r + this->v*t0; // collision point found
		trng = &(srf->polygons[i_coll]); // collision surface found
		n = trng->n;
	}

	// phi - angle of incidence
	double cos_phi = fabs(dot(n,this->v)/(length(n)*length(this->v)));

	// ------------------------------ phys angles & amplitude ---------------------------
	double sin_phi = cos_sin(cos_phi);
	double p = sin_phi / this->c;
	//double p2 = p*p;
	double sin_p, sin_s, cos_p, cos_s;
	double kp, ks; // As = A*ks, Ap = A*kp

	// snell's law
	// TODO sin>1
	Material *mat_from = trng->mat[ bool2int(trng->sg(trng->r[0] - this->r, 0) < 0) ];
	sin_p = p * mat_from->Cp;
	sin_s = p * mat_from->Cs;

	// complex math
	// TODO phys
	switch(this->type){
	case PRayType:
		kp = ks = 0.5;
		break;
	case SRayType:
		kp = ks = 0.5;
		break;
	default:
		error_handl_string = "Wrong ray type: " + toString(this->type) + this->toStr();
		CHECK(WRONG_RAY_TYPE, this->type);
		exit(1);
	}

	// --------------------------------- post-geom - create new rays--------------------------------------
	double3 nr = normalize(cross(n,this->v)); // reflection surface
	double3 v_new;

	// ---- build new P-ray -------
	if(sin_p <= 1){
		cos_p = cos_sin(sin_p);
		v_new = newV(n, this->v, n*cos_p, cross(n, nr)*sin_p) * mat_from->Cp;
		Ray p_ray(rx + v_new * prm->eps, v_new, PRayType, this->A*kp, this->t + t0);
		p_ray.move(srf, prm);
	}
	// Ray(const double3 _r, const double3 _v, const int _type = BaseRayType, const double _A = 1, const double _t = 0):
	// newV(double3 n, double3 v, double3 vx, double3 vy)

	// ---- build new S-ray -------
	if(sin_s <= 1){
		cos_s = cos_sin(sin_s);
		v_new = newV(n, this->v, n*cos_s, cross(n, nr)*sin_s) * mat_from->Cs;
		Ray s_ray(rx + v_new*prm->eps, v_new, SRayType, this->A*ks, this->t + t0);
		s_ray.move(srf, prm);
	}

	prm->n_alive_rays--;
}

pair<double, int> Ray::find_collision(Surface* srf, Params* prm)
{
	int i;
	double t, t_min = prm->Tmax * 2;
	Triangle *trngl;
	int i_coll = -1;

	for(i = 0; i < srf->Npol; ++i){ // find collision point

		trngl = &(srf->polygons[i]); // so we don't have to call [i] every time, also it's shorter
		t = dot(trngl->r[0] - this->r, trngl->n) / dot(this->v, trngl->n); // find time of collision

		if((SYS_EPS < t) && (t < t_min)){
		// if the possible collision can happen (t>0) and if it's better than the one we already have (t<t_min)
			if(trngl->isInside(this->r + this->v*t, prm->eps)){ // if it's really the point, then save it
				t_min = t;
				i_coll = i;
			}
		}
	}

	return make_pair(t_min, i_coll);
}

string Ray::toStr(const string spr)
{
	return spr + (this->type == PRayType ? "P-type" : "S-type") +
			"\nc = " + toString(this->c) + "; A = " + toString(this->A) + "; t = " + toString(this->t) +
			"\nr = " + d3ToStr(this->r) +
			"\nv = " + d3ToStr(this->v) + spr;
}

void Ray::print(ostream &output, string spr)
{
	output << this->toStr(spr);
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Surface -----------------------------------------------
// --------------------------------------------------------------------------------------------

Surface::Surface(int _n_pol, int _n_mat)
{
	this->resize_clear(_n_pol, _n_mat);
}

void Surface::resize_clear(int _n_pol, int _n_mat)
{
	if(_n_pol != this->Npol){
		delete[] this->polygons;
		this->polygons = (_n_pol == 0 ? NULL : (new Triangle[_n_pol]));
		this->Npol = _n_pol;
	}
	if(_n_mat != this->Nmat){
		delete[] this->materials;
		this->materials = (_n_mat == 0 ? NULL : (new Material[_n_mat]));
		this->Nmat = _n_mat;
	}
}

void Surface::print(ostream &output, string spr1, string spr2)
{
	int i;
	output << spr1 << "Nmat = " << this->Nmat << "\n";
	for(i = 0; i < this->Nmat; ++i){
		this->materials[i].print(output, toString(i+1) + ") ");
	}
	output << "Npol = " << this->Npol << "\n";
	for(i = 0; i < this->Npol; ++i){
		this->polygons[i].print(output, "\n------------\n" + toString(i+1) + "\n");
	}
	output << spr2;
}

int Surface::load_from_file(string surf_filename, string mat_filename)
{
	ifstream input;
	int i, i2, i3, i4, i5;
	string buf_str;

	// --------------------------------------- read materials -----------------------------------------------
	input.open(mat_filename);
	if(!input){
		input.close();
		error_handl_string = "material-file " + mat_filename + " is missing\n";
		return CANT_OPEN_FILE_FOR_READING;
	}

	int _n_mat;
	input >> _n_mat;
	if(_n_mat < 2){
		error_handl_string = "Nmat = " + toString(_n_mat) + "; must be at least 2\n";
		return LESS_2_MATERIALS;
	}
	this->resize_clear(this->Npol, _n_mat);
	for(i = 0; i < this->Nmat; ++i){
		input >> this->materials[i].Cs >> this->materials[i].Cp;
	}
	input.close();


	// ---------------------------------------------------------------------------------------------------
	// ------------------------------------- read surface ------------------------------------------------
	// ---------------------------------------------------------------------------------------------------
	input.open(surf_filename);
	if(!input){
		input.close();
		error_handl_string = "surface-file " + surf_filename + " is missing\n";
		return CANT_OPEN_FILE_FOR_READING;
	}
	double3 *vertex;
	int Nvertex;
	// --------------------------------------- read vertex -----------------------------------------------
	input >> Nvertex;
	vertex = new double3[Nvertex];
	for(i = 0; i < Nvertex; ++i){
		input >> vertex[i].x >> vertex[i].y >> vertex[i].z;
	}

	// ------------------------------------- read tetrahedrons & triangles ---------------------------------------------
	int Ntet, i_mat, i_vertex[4];

	input >> Ntet;

	vector<Triangle> read_polygons;
	int n_overlap_vrtx, i_curr_mat;
	int tet_surf_ind[4][3] =
	// tet_surf_ind[i][:] - all vertices except i-th one -
	// the indices of the i-th triangle-sub-surf of the abstract tetrahedron
	{
	  {1, 2, 3},
	  {0, 2, 3},
	  {0, 1, 3},
	  {0, 1, 2}
	};

	for(i = 0; i < Ntet; ++i){
		for(i2 = 0; i2 < 4; ++i2){ // read vertices of tetrahedron_i
			input >> i_vertex[i2];
			--i_vertex[i2]; // arrays from 0, in file-format from 1
		}
		input >> i_mat; // read material of tetrahedron_i

		for(i2 = 0; i2 < 4; ++i2){           // for each tetrahedron sub-surface (tet_surf_ind[i2][:]) check overlaps
			for(i3 = 0; i3 < read_polygons.size(); ++i3){       // with all existing ones
				n_overlap_vrtx = 0;
				for(i4 = 0; i4 < 3; ++i4){
					for(i5 = 0; i5 < 3; ++i5){
						/*
						 * i - index of the current tetrahedron
						 * i2 - index of surface of current tetrahedron
						 * i3 - for all existing sufraces - index of some existing surface to check a pair tetr_surf<->existing_surf
						 * i4 - index of the vertex of the current surf of the current tetr
						 * i5 - index of the vertex of the current existing triangle
						 */
						if(i_vertex[ tet_surf_ind[i2][i4] ] == read_polygons[i3].i_vrtx[i5]){
							++n_overlap_vrtx;
							break;
						}
					}
				}
				if(n_overlap_vrtx >= 3){
					break;
					/*
					 * current (i2) triangle-sub-surf can't overlap with more than 1 existing triangle.
					 * It's found, so no overlapping with all further ones.
					 * */
				}
			}
			if(n_overlap_vrtx > 3){
				error_handl_string = "n_overlap_vrtx = " + toString(n_overlap_vrtx) + "\n i = " + toString(i) + "; i2 = " + toString(i2) + "; i3 = " + toString(i3);
				delete[] vertex;
				read_polygons.clear();
				input.close();
				return TOO_MANY_OVERLAPED_VERTICES;
			}
			if(i3 == read_polygons.size()){ // current (i2) triangle-sub-surf is new to the all existing ones
				read_polygons.push_back(Triangle(vertex, i_vertex[ tet_surf_ind[i2][0] ],
						                                 i_vertex[ tet_surf_ind[i2][1] ],
						                                 i_vertex[ tet_surf_ind[i2][2] ]));
			}
			// else means already used polygons similar to the current (i2) one was found
			// read_polygons[i3].i_vrtx[:] == i_vertex[ tet_surf_ind[i2][:] ]

			/*
			 * if the current triangle is a new one, the i3 is the number of polygons BEFORE new one was added,
			 * so after all i3 is the index of the last added triangle. In this case i3 == read_polygons.size()-1
			 *
			 * otherwise (existing similar one was found) i3 is the index of the triangle identical to the current one.
			 * In this case we add 2nd matetrial to the existing polygon and don't add a new triangle to polygons
			 *
			 * So in both cases i3 is the index of the triangle to add the current material to
			 */
			i_curr_mat = bool2int(read_polygons[i3].sg(vertex[ i_vertex[i2] ] - read_polygons[i3].r[0], 0) > 0);
			// i2 is the missing index in tet_surf_ind[i2][:] set, so i_vertex[i2] is the 4th vertex of the current tetrahedron
			read_polygons[i3].mat[i_curr_mat] = this->materials + i_mat;

			if(read_polygons[i3].mat[i_curr_mat]->isEq(read_polygons[i3].mat[bool2int(!i_curr_mat)])){
				// read_polygons[i3] is a formal border with no real meaning
				read_polygons.erase(read_polygons.begin()+i3);
			}
		}
	}
	/*
	 * As a result we have read_polygons with all true-surface triangles of the system
	 * Now we need to add background material and copy read polygons to the main array
	 */
	for(i = 0; i < read_polygons.size(); ++i){ // if some materials are unset, then set them to background material
		for(i2 = 0; i2 < 2; ++i2){
			if(read_polygons[i].mat[i2] == NULL){
				read_polygons[i].mat[i2] = this->materials; // &(material[0])
			}
		}
		if(read_polygons[i].mat[0]->isEq(read_polygons[i].mat[1])){
			// read_polygons[i] is a formal border with no real meaning
			// actually it can't happen here if everything before worked fine, but lets leave it this way for now
			// TODO error polygon with no material
			read_polygons.erase(read_polygons.begin()+i);
		}
	}

	this->resize_clear(read_polygons.size(), this->Nmat);
	for(i = 0; i < this->Npol; ++i){ // copy read materials
		this->polygons[i] = read_polygons[i];
	}

	delete[] vertex;
	read_polygons.clear();
	input.close();
	return 0;
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Surface -----------------------------------------------
// --------------------------------------------------------------------------------------------


int Params::load_from_file(string filename)
{
	ifstream input(filename);
	if(!input){
		input.close();
		return CANT_OPEN_FILE_FOR_READING;
	}

	string buf_s;
	std::getline(input, buf_s); // read comment line

	double _eps;
	input >> this->Nrays >> this->Tmax >> this->Amin >> _eps;
	this->eps = _eps > 0 ? _eps : SYS_EPS;
	input.close();

	return 0;
}

void Params::print(ostream &output, string spr)
{
	for(int i = 0; i < this->paramFHead.size(); ++i){
		output << this->paramFHead[i] << " | ";
	}
	output << spr << this->Nrays << spr
		   << this->Tmax << spr
		   << this->Amin << spr
		   << this->eps << spr
		   << "model_name: |" << this->model_name << "|" << spr
		   << "alive_rays: " << this->n_alive_rays << "; total_rays: " << this->n_total_rays << spr;
}

int Params::save_to_file(string filename)
{
	ofstream output(filename);
	if(!output){
		return CANT_OPEN_FILE_FOR_WRITING;
	}

	this->print(output);

	output.close();
	return 0;
}
