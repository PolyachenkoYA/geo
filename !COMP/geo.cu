#include "hip/hip_runtime.h"

/**
 * PolyachYA Corporation.  All rights reserved.
 *
 * Please refer to the PolyachYA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include "geo.cuh"

string error_handl_string;

// ----------------------------------------- main math ------------------------------------------

int rightV(vector<double3>& v, double3 n, double3 v0)
{
	if(v.size() != 4){
		CHECK(WRONG_SET_OF_POSSIBLE_REFLECTED_RAYS, 0);
		exit(1);
	}


	int i,res = -1;
	double cs, max_cs = -2, kv = 1/length(v0);

	// reflected ray have the least angle with v0 |=> the beggest cos(v_new,v0)
	for(i = 0; i < 4; ++i){
		if(dot(v0, n)*dot(v[i], n) < 0){ // determine other side of surface
			cs = dot(v[i], v0)/length(v[i])*kv;
			if(max_cs < cs){ // determine reflected ray
				res = i;
				max_cs = cs;
			}
		}
	}

	if(res == -1){
		CHECK(NO_REFLECTED_RAY_FOUND, 0);
		exit(1);
	}
	return res;
}

double3 newV(double3 n, double3 v_old, double3 vx, double3 vy)
{
	vector<double3> v_news;
	v_news.clear();
	// we don't know the direction of n and nr, so we have to check all 4 possible variants
	// to choose the one that really is the physicaly reflected ray
	v_news.push_back(vx + vy);
	v_news.push_back(vx - vy);
	v_news.push_back(-v_news[1]);
	v_news.push_back(-v_news[0]);
	return v_news[rightV(v_news, n, v_old)];
}

// ----------------------------------------- Other stuff ----------------------------------------------
template<typename T>
void stp(T str)
{
	cout << str << endl; cin.get();
}

template<typename T>
string vectorToStr(vector<T> v, string sp)
{
        string s="";
        for(int i = 0; i < v.size()-1; ++i) s+= (toString(v[i])+sp);
        return s+toString(v[v.size()-1]);
}
template<typename T>
void printVector(ostream &output, vector<T> v, string sp1, string sp2, string sp3)
{
	int sz = v.size();
    output << sp1;
    for(int i = 0; i < sz-1; ++i) output << v[i] << sp2;
    output << v[sz-1] << sp3;
}
template<typename T>
T sumVector(vector<T> v)
{
        T s=0;
        for(int i = 0; i < v.size(); ++i) s+=v[i];
        return s;
}
template <typename T>
string toString(T val)
{
    std::ostringstream oss;
    oss << val;
    return oss.str();
}

template<typename T>
T fromString(const string& s)
{
  std::istringstream iss(s);
  T res;
  iss >> res;
  return res;
}

string toLower(string s)
{
	char d = 'a'-'A';
	for(int i = 0; i<s.size(); ++i) if((s[i]>='A') && (s[i]<='Z')) s[i]+=d;
	return s;
}
string toUpper(string s)
{
	char d = 'A'-'a';
	for(int i = 0; i<s.size(); ++i) if((s[i]>='a') && (s[i]<='z')) s[i]+=d;
	return s;
}
void printD3(ostream &output, double3 r, string sp1, string sp2, string sp3)
{
	output << sp1 << r.x << sp2 << r.y << sp2 << r.z << sp3;
}

int find_partition(ifstream &input, string mask, bool whole_file = 0)
{
	if(whole_file){
		input.clear();
		input.seekg(0, ios::beg);
	}
	int res = 0;

	char buf_chars[256];
	string buf_str;
	do{
		++res;
		input.getline(buf_chars, 255);
		buf_str = string(buf_chars);
		// getline doesn't remove '\n' in the end of a line, so we compare mask with buf_str[0:-1]
	}while(buf_str.compare(0, buf_str.size()-1, mask) && !input.eof());

	if(buf_str.compare(0, buf_str.size()-1, mask)){
		CHECK(WRONG_MSH_FILE_ID, "No |" + mask + "| partition found in msh file");
		exit(1);
	}

	return res;
}

template<typename T>
int CHECK(int n, T s, string logFname)
{
    if(!n){ return 0; }
    ofstream Fout(logFname.c_str(), ios::app);
    if(!Fout){ return CANT_OPEN_FILE_FOR_WRITING; }

    if(n != SAY_IT){
    	Fout << "\nerror #" << toString(n) << "\n";
    	cout << "error #" << toString(n) << "\n";
    }
    switch(n){
    	case SAY_IT: Fout << s; break;

    	case CANT_OPEN_FILE_FOR_READING: Fout << "Can't open file\n" << s << "\nfor reading"; break;
    	case CANT_OPEN_FILE_FOR_WRITING: Fout << "Can't open file\n" << s << "\nfor writing"; break;

    	case TOO_BIG_NODE_IND: Fout << "Too big node ID in file\n" << s << "\n" << error_handl_string; break;
    	case TOO_BIG_ELEMENT_IND: Fout << "Too big element ID in file\n" << s << "\n" << error_handl_string; break;
    	case WRONG_MSH_FILE_ID: Fout << s; break;

    	case WRONG_SET_OF_POSSIBLE_REFLECTED_RAYS: Fout << "Wrong set of possible reflected rays (v.size != 4)\n"; break;
    	case NO_REFLECTED_RAY_FOUND: Fout << "No reflected ray found\n"; break;

        default: Fout << "Unknown error #" << n;
    }
    if(n != SAY_IT){ Fout << "\n"; }
    Fout.close();

    return n;
}

template<typename T>
void SAY_LOG(T s, string logFname)
{
	CHECK(SAY_IT, s, logFname);
}

// ---------------------------------------- Other math -------------------------------------------------
bool almostEq(double x, double y, double _eps)
{
	return y == 0 ? (abs(x) < _eps) : (abs(x/y - 1) < _eps);
}

vector<double> d3ToV(double3 v){
	double vp[3] = {v.x, v.y, v.z};
	vector<double> vv;
	vv.assign(vp, vp+3);
	return vv;
}

double cos_sin(double x)
{
    return sqrtf(1-x*x);
}

double sqr(double x)
{
	return x*x;
}

int sgn(double x, double _eps)
{
    if(x > _eps){
    	return 1;
    } else if(x < -_eps){
        return -1;
    } else {
        return 0;
    }
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Triangle ----------------------------------------------
// --------------------------------------------------------------------------------------------
double3 Triangle::getNorm(void) const
{
    return normalize(cross(this->r[1] - this->r[0], this->r[2] - this->r[0]));
}

int Triangle::sg(double3 rx) const
{
    return sgn(dot(rx, this->n));
}

int Triangle::isInside(double3 rx) const
{
    int sg0 = this->sg(cross(this->r[1] - rx, this->r[1] - this->r[0]));
    if(sg0 == 0){
        return 1;
    } else{
        return (sg0 == this->sg(cross(this->r[2] - rx, this->r[2] - this->r[1]))) &&
               (sg0 == this->sg(cross(this->r[0] - rx, this->r[0] - this->r[2])));
    }
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Ray ---------------------------------------------------
// --------------------------------------------------------------------------------------------

void Ray::move(Surface& srf)
{
	this->print(cout);
	prm.n_alive_rays++;
	prm.n_total_rays++;

	if((this->t + length(this->r)/this->c > prm.Tmax) || (this->A < prm.Amin)){
		prm.n_alive_rays--;
		return;
	}

	//int i;

	// --------------------- pre-geom - find collision point -------------------------
	double3 n, rx;
	pair<double, int> coll_res = this->find_collision(srf);
	double t0 = coll_res.first;
	int i_coll = coll_res.second;
	Triangle trng;

	if(i_coll == -1){
		prm.n_alive_rays--;
		return;
	} else {
		rx = this->r + this->v*t0; // collision point found
		trng = srf.polygons[i_coll]; // collision surface found
		n = trng.n;
	}

	// phi - angle of incidence
	double cos_phi = fabs(dot(n,this->v)/(length(n)*length(this->v)));

	// ------------------------------ phys angles & amplitude ---------------------------
	double sin_phi = cos_sin(cos_phi);
	double p = sin_phi / this->c;
	//double p2 = p*p;
	double sin_p, sin_s, cos_p, cos_s;
	double kp, ks; // As = A*ks, Ap = A*kp

	// snell's law
	// TODO sin>1
	sin_p = p * prm.Cp;
	sin_s = p * prm.Cs;

	// complex math
	// TODO phys
	switch(this->type){
	case PRayType:
		kp = ks = 0.5;
		break;
	case SRayType:
		kp = ks = 0.5;
		break;
	default:
		cerr << "wrong ray type\n";
	}

	// --------------------------------- post-geom - create new rays--------------------------------------
	double3 nr = normalize(cross(n,this->v)); // reflection surface
	double3 v_new;

	// ---- build new P-ray -------
	if(sin_p <= 1){
		cos_p = cos_sin(sin_p);
		v_new = newV(n, this->v, n*cos_p, cross(n, nr)*sin_p);
		Ray p_ray(rx + v_new*prm.eps, v_new, PRayType, this->A*kp, this->t + t0);
		p_ray.move(srf);
	}
	// Ray(const double3 _r, const double3 _ve, const int _type = PRayType, const double _A = 1, const double _t = 0)
	// newV(double3 n, double3 v, double3 vx, double3 vy)

	// ---- build new S-ray -------
	if(sin_s <= 1){
		cos_s = cos_sin(sin_s);
		v_new = newV(n, this->v, n*cos_s, cross(n, nr)*sin_s);
		Ray s_ray(rx + v_new*prm.eps, v_new, SRayType, this->A*ks, this->t + t0);
		s_ray.move(srf);
	}

	prm.n_alive_rays--;
}

pair<double, int> Ray::find_collision(Surface& srf)
{
	int i;
	double t, t_min = prm.Tmax*2;
	Triangle *trngl;
	int i_coll = -1;

	for(i = 0; i < srf.size; ++i){ // find collision point

		trngl = &(srf.polygons[i]); // so we don't have to call [i] every time, also it's shorter
		t = dot(trngl->r[0] - this->r, trngl->n) / dot(this->v, trngl->n); // find time of collision

		if((0 < t) && (t < t_min)){ // if the possible collision can happen (t>0) and if better than the one we already have(t<t_min)
			if(trngl->isInside(this->r + this->v*t)){ // if it's really the point, then save it
				t_min = t;
				i_coll = i;
			}
		}
	}

	return make_pair(t_min, i_coll);
}

void Ray::print(ostream &output, string spr)
{
	output << spr;
	output << (this->type == PRayType ? "P-type" : "S-type") << "\n";
	output << "c = " << this->c << "; A = " << this->A << "; t = " << this->t << "\n";
	output << "r = ";
	printD3(output, this->r);
	output << "v = ";
	printD3(output, this->v);
	output << spr;
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Surface -----------------------------------------------
// --------------------------------------------------------------------------------------------

Surface::Surface(int _n)
{
	if(_n == 0){
		this->polygons = NULL;
		this->size = 0;
	} else {
		this->resize_clear(_n);
	}
}

void Surface::resize_clear(int _n)
{
	if(this->polygons != NULL){
		delete[] this->polygons;
	}
	if(_n == 0)
		return;

	this->polygons = new Triangle[_n];
	this->size = _n;
}

int Surface::load_from_file(string filename)
{
	ifstream input(filename);
	if(!input){
		input.close();
		return CANT_OPEN_FILE_FOR_READING;
	}

	int i, err_handl;
	double buf_d;
	string buf_str;
	double3 *nodes;
	int Nnodes, k;

	int node_partition_i = find_partition(input, "$Nodes", 0);
	input >> Nnodes;
	nodes = new double3[Nnodes];
	for(i = 0; i < Nnodes; ++i){ // load nodes
		input >> k;
		--k; // k begins from 1, array begins from 0
		// actually the .gmh format says that nodes don't have to be ordered, so it's possible for 'k' to be > Nnodes.
		// But for our work lets state (at least for now) that it can't be so.
		// All nodes [1;N] have to be set before N+1 node will be set
		if(k >= Nnodes){
			delete[] nodes;
			input.close();
			error_handl_string = "local_line_ind = " + toString(i) + "; k = " + toString(k+1);
			return TOO_BIG_NODE_IND;
		}
		input >> nodes[k].x >> nodes[k].y >> nodes[k].z;
	}

	int elements_partition_i = find_partition(input, "$Elements", 0);
	int Nel, elType, k_r1, k_r2, k_r3;
	input >> Nel;
	this->resize_clear(Nel);
	for(i = 0; i < this->size; ++i){ // build elements using already read nodes
		input >> k;
		--k; // k begins from 1, array begins from 0
		if(k >= this->size){
			delete[] nodes;
			input.close();
			error_handl_string = "local_line_ind = " + toString(i) + "; k = " + toString(k+1);
			return TOO_BIG_ELEMENT_IND;
		}

		input >> elType;

		if(elType == 2){
			input >> buf_d; // teg_number
			input >> buf_d; // 1st teg - number of the physical entity to which the element belongs
			input >> buf_d; // 2nd teg - number of the elementary geometrical entity to which the element belongs

			input >> k_r1 >> k_r2 >> k_r3;
			// k begins from 1, array begins from 0
			--k_r1;
			--k_r2;
			--k_r3;
			if((k_r1 >= Nnodes) || (k_r2 >= Nnodes) || (k_r3 >= Nnodes)){
				delete[] nodes;
				input.close();
				error_handl_string = "local_line_ind = " + toString(i) + "; k1 = " + toString(k_r1+1) + "; k2 = " + toString(k_r2+1) + "; k3 = " + toString(k_r3+1);
				return TOO_BIG_NODE_IND;
			}

			this->polygons[k] = Triangle(nodes[k_r1], nodes[k_r2], nodes[k_r3]);
		} else {
			// kind of error, but ok, let's just warn the user about not complitely supported format
			buf_str = "line " + toString(i) + " in file\n" + filename + "\nis't a triangle\nOnly triangles are supported yet.\n";
			SAY_LOG(buf_str);
			cout << buf_str;
		}
	}

	delete[] nodes;
	input.close();
	return 0;
}

// --------------------------------------------------------------------------------------------
// ------------------------------------ Surface -----------------------------------------------
// --------------------------------------------------------------------------------------------


int Params::load_from_file(string filename)
{
	ifstream input(filename);
	if(!input){
		input.close();
		return CANT_OPEN_FILE_FOR_READING;
	}

	double _eps;
	input >> this->Nrays >> this->Tmax >> this->Amin >> this->Cp >> this->Cs >> _eps;
	this->eps = _eps > 0 ? _eps : SYS_EPS;

	if(!input.eof()){
		char buf_str[256];
		input.getline(buf_str, 255);
		this->msh_filename = string(buf_str);
	}

	input.close();
	return 0;
}

int Params::print(ostream &output, string spr)
{
	output << this->Nrays << spr
		   << this->Tmax << spr
		   << this->Amin << spr
		   << this->Cp << spr
		   << this->Cs << spr
		   << this->eps << spr
		   << this->msh_filename << spr;

	return 0;
}

int Params::save_to_file(string filename)
{
	ofstream output(filename);
	if(!output){
		return CANT_OPEN_FILE_FOR_WRITING;
	}

	this->print(output);

	output.close();
	return 0;
}
