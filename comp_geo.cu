#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : comp_geo.cu
 Author      : PolyachYA
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include "../../geo.cu"

int main(void)
{
	double h = 10;
	double big_dist = 2*MapSize;

	// ------------ create test surface --------------
	Surf surf;
	Triangle trng;
	double3 r1,r2,r3;
	// upper bound
	r1 = make_double3(-big_dist, h, big_dist);
	r2 = make_double3(-big_dist, h, -big_dist);
	r3 = make_double3(big_dist, h, big_dist);
	trng = Triangle(r1,r2,r3);
	surf.add(trng);
	r1 = make_double3(-big_dist, h, -big_dist);
	r2 = make_double3(big_dist, h, -big_dist);
	r3 = make_double3(big_dist, h, big_dist);
	trng = Triangle(r1,r2,r3);
	surf.add(trng);

	// lower bound
	r1 = make_double3(-big_dist, -h, big_dist);
	r2 = make_double3(-big_dist, -h, -big_dist);
	r3 = make_double3(big_dist, -h, big_dist);
	trng = Triangle(r1,r2,r3);
	surf.add(trng);
	r1 = make_double3(-big_dist, -h, -big_dist);
	r2 = make_double3(big_dist, -h, -big_dist);
	r3 = make_double3(big_dist, -h, big_dist);
	trng = Triangle(r1,r2,r3);
	surf.add(trng);

	// ------------ create test rays ---------------
	Wavefront waves;
	Ray ray(make_double3(0,0,0), normalize(make_double3(1,2,0)));
	waves.add(ray);

	// ------------- conduct test computation -------
	//move_ray(ray, surf);
	ray.move(surf);

	return 0;
}

