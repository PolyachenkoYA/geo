#include "hip/hip_runtime.h"

/**
 * PolyachYA Corporation.  All rights reserved.
 *
 * Please refer to the PolyachYA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include "geo.cuh"

long long int n_rays = 0;
long long int n_tot = 0;

// ----------------------------------------- main math-body ------------------------------------------

int rightV(vector<double3>& v, double3 n, double3 v0)
{
	if(v.size() != 4)
		return -1;

	int i,res = -1;
	double cs, max_cs = -2, kv = 1/length(v0);

	for(i = 0; i < 4; ++i){
		if(dot(v0, n)*dot(v[i], n) < 0){ // determine other side of surface
			cs = dot(v[i], v0)/length(v[i])*kv;
			if(max_cs < cs){ // determine reflected ray
				res = i;
				max_cs = cs;
			}
		}
	}

	if(res == -1){
		cerr << "No prover V found (rightV)\n";
		exit(1);
	}
	return res;
}

double3 newV(double3 n, double3 v_old, double3 vx, double3 vy)
{
	vector<double3> v_news;
	v_news.clear();
	v_news.push_back(vx + vy);
	v_news.push_back(vx - vy);
	v_news.push_back(-v_news[1]);
	v_news.push_back(-v_news[0]);
	return v_news[rightV(v_news, n, v_old)];
}

// ----------------------------------------- Other stuff ----------------------------------------------
template<typename T>
void stp(T str){ cout << str << endl; cin.get(); }

template<typename T>
string vectorToStr(vector<T> v, string sp)
{
        string s="";
        for(int i = 0; i < v.size()-1; ++i) s+= (toString(v[i])+sp);
        return s+toString(v[v.size()-1]);
}
template<typename T>
void printVector(ostream &output, vector<T> v, string sp1, string sp2, string sp3)
{
	int sz = v.size();
    output << sp1;
    for(int i = 0; i < sz-1; ++i) output << v[i] << sp2;
    output << v[sz-1] << sp3;
}
template<typename T>
T sumVector(vector<T> v)
{
        T s=0;
        for(int i = 0; i < v.size(); ++i) s+=v[i];
        return s;
}
template <typename T>
string toString(T val)
{
    std::ostringstream oss;
    oss << val;
    return oss.str();
}

template<typename T>
T fromString(const string& s)
{
  std::istringstream iss(s);
  T res;
  iss >> res;
  return res;
}

string toLower(string s)
{
	char d = 'a'-'A';
	for(int i = 0; i<s.size(); ++i) if((s[i]>='A') && (s[i]<='Z')) s[i]+=d;
	return s;
}
string toUpper(string s)
{
	char d = 'A'-'a';
	for(int i = 0; i<s.size(); ++i) if((s[i]>='a') && (s[i]<='z')) s[i]+=d;
	return s;
}
void printD3(ostream &output, double3 r, string sp1, string sp2, string sp3)
{
	output << sp1 << r.x << sp2 << r.y << sp2 << r.z << sp3;
}


// ---------------------------------------- Other math -------------------------------------------------
bool almostEq(double x, double y, double _eps)
{
	return y == 0 ? (abs(x) < _eps) : (abs(x/y - 1) < _eps);
}

vector<double> d3ToV(double3 v){
	double vp[3] = {v.x, v.y, v.z};
	vector<double> vv;
	vv.assign(vp, vp+3);
	return vv;
}

double cos_sin(double x)
{
    return sqrtf(1-x*x);
}

double sqr(double x)
{
	return x*x;
}

int sgn(double x, double _eps)
{
    if(x > _eps){
    	return 1;
    } else if(x < -_eps){
        return -1;
    } else {
        return 0;
    }
}

// ------------------------------------ Triangle ----------------------------------------------
double3 Triangle::getNorm(void) const
{
    return normalize(cross(this->r[1] - this->r[0], this->r[2] - this->r[0]));
}

int Triangle::sg(double3 rx) const
{
    return sgn(dot(rx, this->n));
}

int Triangle::isInside(double3 rx) const
{
    int sg0 = this->sg(cross(this->r[1] - rx, this->r[1] - this->r[0]));
    if(sg0 == 0){
        return 1;
    } else{
        return (sg0 == this->sg(cross(this->r[2] - rx, this->r[2] - this->r[1]))) &&
               (sg0 == this->sg(cross(this->r[0] - rx, this->r[0] - this->r[2])));
    }
}

// -------------------------------------- Ray --------------------------------------------------

void Ray::move(Surf& srf)
{
	n_rays++;
	n_tot++;

	if((this->t + length(this->r)/this->c > Tmax) || (this->A < Amin)){
		n_rays--;
		return;
	}

	//int i;

	// --------------------- pre-geom - find collision point -------------------------
	double3 n, rx;
	pair<double, int> coll_res = this->find_collision(srf);
	double t0 = coll_res.first;
	int i_coll = coll_res.second;
	Triangle trng;

	if(i_coll == -1){
		n_rays--;
		return;
	} else {
		rx = this->r + this->v*t0; // collision point found
		trng = srf.polygons[i_coll]; // collision surface found
		n = trng.n;
	}

	// phi - angle of incidence
	double cos_phi = fabs(dot(n,this->v)/(length(n)*length(this->v)));

	// ------------------------------ phys angles & amplitude ---------------------------
	double sin_phi = cos_sin(cos_phi);
	double p = sin_phi / this->c;
	//double p2 = p*p;
	double sin_p, sin_s, cos_p, cos_s;
	double kp, ks; // As = A*ks, Ap = A*kp

	// snell's law
	// TODO sin>1
	sin_p = p*Cp;
	sin_s = p*Cs;

	// complex math
	// TODO phys
	switch(this->type){
	case PRayType:
		kp = ks = 0.5;
		break;
	case SRayType:
		kp = ks = 0.5;
		break;
	default:
		cerr << "wrong ray type\n";
	}

	// --------------------------------- post-geom - create new rays--------------------------------------
	double3 nr = normalize(cross(n,this->v)); // reflection surface
	double3 v_new;

	// ---- build new P-ray -------
	if(sin_p <= 1){
		cos_p = cos_sin(sin_p);
		v_new = newV(n, this->v, n*cos_p, cross(n, nr)*sin_p);
		Ray p_ray(rx + v_new*eps, v_new, PRayType, this->A*kp, this->t + t0);
		p_ray.move(srf);
	}
	// Ray(const double3 _r, const double3 _ve, const int _type = PRayType, const double _A = 1, const double _t = 0)
	// newV(double3 n, double3 v, double3 vx, double3 vy)

	// ---- build new S-ray -------
	if(sin_s <= 1){
		cos_s = cos_sin(sin_s);
		v_new = newV(n, this->v, n*cos_s, cross(n, nr)*sin_s);
		Ray s_ray(rx + v_new*eps, v_new, SRayType, this->A*ks, this->t + t0);
		s_ray.move(srf);
	}

	n_rays--;
}

pair<double, int> Ray::find_collision(Surf& srf)
{
	int i;
	double t, t_min = Tmax*2;
	Triangle *trngl;
	int i_coll = -1;

	for(i = 0; i < srf.polygons.size(); ++i){ // find collision point

		trngl = &(srf.polygons[i]); // so we don't have to call [i] every time, also it's shorter
		t = dot(trngl->r[0] - this->r, trngl->n) / dot(this->v, trngl->n); // find time of collision

		if((0 < t) && (t < t_min)){ // if the possible collision can happen (t>0) and if better than the one we already have(t<t_min)
			if(trngl->isInside(this->r + this->v*t)){ // if it's really the point, then save it
				t_min = t;
				i_coll = i;
			}
		}
	}

	return make_pair(t_min, i_coll);
}

void Ray::print(ostream &output, string spr)
{
	output << spr;
	output << (this->type == PRayType ? "P-type" : "S-type") << "\n";
	output << "c = " << this->c << "; A = " << this->A << "; t = " << this->t << "\n";
	output << "r = ";
	printD3(output, this->r);
	output << "v = ";
	printD3(output, this->v);
	output << spr;
}

